#include "hip/hip_runtime.h"
/* SST.cu

GEOSTATISTICAL APPLICATION "v2_1" uses checkerboard spin-flip Metropolis simulation
of a two-dimensional ferromagnetic XY model with modified hamiltonian
H = -J sum_{ij} cos( Qfactor*(theta_i - theta_j) )
on graphics processing units (GPUs) using the NVIDIA CUDA framework.

Implements spatially variable MPR using single checkerboard decomposition and site-specific temperatures.
*/

#ifdef __unix
#define fopen_s(pFile,filename,mode) ((*(pFile))=fopen((filename),(mode)))==NULL
#endif


#include <iostream>
#include <fstream>
//#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <cstdio>
#define _USE_MATH_DEFINES	// for pi constant
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <hip/device_functions.h> internal header, deprecated in the future, misov povodny kod
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <vector>
//#include <list>
#include <iterator>
#include <algorithm> 		// for sort operation
#include <limits>			// has own min(), max() ???? which behave correctly with NAN (ignoing NAN values)
#include <windows.h>
#include <random>

#include <hipcub/hipcub.hpp>

#undef min
#undef max

#include <chrono>	// high precision execution time measurment

/*#include <thread>*/
//#include <boost/timer/timer.hpp>

//using namespace std;

#define DIM 2

#define L 256    // minumum of L = 2*BLOCKL

#define Qfactor 0.5f	// uncomment for single precision calculations in kernel
#define BLOCKL 32
#define GRIDL (L/BLOCKL)
#define BLOCKS ((GRIDL*GRIDL)/2)
#define THREADS ((BLOCKL*BLOCKL)/2)
#define N (L*L)
#define Nbond (2*L*(L - 1))
//#define TOTTHREADS (BLOCKS*THREADS)



#define SWEEPS_GLOBAL 100
#define SWEEPS_EMPTY 1
#define CONFIG_SAMPLES 100		// M = 100


#define ACC_RATE_MIN_EQ 0.30		// A_targ = 0.3
#define ACC_RATE_MIN_SIM 0.30		


//#define ACC_TEST_FREQUENCY_EQ 10	//ACC_TEST_FREQUENCY_EQ 
//#define ACC_TEST_FREQUENCY_SIM 10	//ACC_TEST_FREQUENCY_SIM
#define EQUI_TEST_FREQUENCY 5   // n_f = 5
#define EQUI_TEST_SAMPLES 20    // n_fit povodne 20



#define SWEEPS_EQUI_MAX 300		// upper limit for equilibration hybrid sweeps; probably not necessary
#define SLOPE_RESTR_FACTOR 3.0	// k_a = 3; for a = 1 + i/k_a (SLOPE_RESTR = k_a)
#define SMOOTHING_PARAMETER 0  //number of times the smoothing procedure is carried out

#define RemovedDataRatio 0.8f


//#define SOURCE_DATA_PATH "zo_L2048_ka02_nu05.bin"
#define SOURCE_DATA_PATH "walker_lake.bin"
//#define SOURCE_DATA_PATH "wall_3_L2048.bin"
#define SOURCE_DATA_NAME "walker"
/*#define RNG_SEED_DILUTION 1564564ULL
#define RNG_SEED_FILL 25756655ULL
#define RNG_SEED_SIMULATION 3456454624ULL*/
#define RNG_SEED_DILUTION 842301111UL
#define RNG_SEED_FILL 5451UL
#define RNG_SEED_SIMULATION 3645445443UL

//#define DOUBLE_PRECISION
#ifndef DOUBLE_PRECISION
#define INTRINSIC_FLOAT
#endif

#define OVER_RELAXATION_EQ
//#define OVER_RELAXATION_SIM

//comment these for time measurements
#define ENERGIES_PRINT
//#define CONFIGURATION_PRINT
//#define RECONSTRUCTION_PRINT
//#define ERROR_PRINT
//#define TEMPSnALPHAS_PRINT
//#define RANDOM_INIT
//#define DIL_ENERGIES_PRINT	// not working yet

//#define SOURCE_MAPPING

//#define COLD_START			// not working yet

// other macros
// linear congruential generator
#define AA 1664525
#define CC 1013904223
#define RAN(n) (n = AA*n + CC)
#define MULT 2.328306437080797e-10f
/*
#define MULT2 4.6566128752457969e-10f
*/
#define sS(x,y) sS[(y+1)*(BLOCKL+2)+x+1]

typedef double source_t;
#ifdef DOUBLE_PRECISION
typedef double spin_t;
typedef double energy_t;
#else
typedef float spin_t;
typedef float energy_t;
#endif

// GPU processing partition
const dim3 gridLinearLattice((int)ceil(N / 256.0));
const dim3 gridLinearLatticeHalf((int)ceil(N / 2.0 / 256.0));
const dim3 blockLinearLattice(256);

//pre double checkerboard
const dim3 grid(GRIDL, GRIDL / 2);
const dim3 block(BLOCKL, BLOCKL / 2);
//pre single checkerboard
const dim3 grid_check(GRIDL, GRIDL);
const dim3 block_check(BLOCKL, BLOCKL / 2);

//hotfixed * 2
const dim3 gridAcc((int)ceil(BLOCKS * 2 / 128.0));
const dim3 blockAcc(128);

const dim3 gridEn(GRIDL, GRIDL);
const dim3 blockEn(BLOCKL, BLOCKL);


// CUDA error checking macro
#define CUDAErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s ; %s ; line %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// cuRAND error checking macro
#define cuRAND_ErrChk(err) { if (err != HIPRAND_STATUS_SUCCESS) std::cout << curandGetErrorString(err) << "\n"; }

// cuRAND errors
char* curandGetErrorString(hiprandStatus_t);
const char* curanderr[14] = {
    "No errors", "Header file and linked library version do not match",
    "Generator not initialized", "Memory allocation failed",
    "Generator is wrong type", "Argument out of range",
    "Length requested is not a multiple of dimension",
    "GPU does not have double precision required by MRG32k3a",
    "Kernel launch failure", "Pre-existing failure on library entry",
    "Initialization of CUDA failed", "Architecture mismatch, GPU does not support requested feature",
    "Internal library error", "Unknown error"
};


//single checkerboard
__global__ void metro_conditioned_sublattice_k(spin_t*, spin_t*, float*, unsigned int, energy_t*, unsigned int*, spin_t*);
__global__ void spin_mult(spin_t*, spin_t);
__global__ void over_relaxation_k(spin_t*, spin_t*, int);
__global__ void energyCalc_k(spin_t*, energy_t*);
__global__ void energyCalcDiluted_k(spin_t*, energy_t*);
__global__ void energyCalcDiluted_per_block(energy_t*, unsigned int*, unsigned int);
__global__ void min_max_avg_block(spin_t*, spin_t*, spin_t*, spin_t*);
__global__ void resetAccD_k(energy_t*);
__global__ void min_max_k(source_t*, source_t*, source_t*, bool, spin_t*);
__global__ void XY_mapping_k(source_t*, spin_t*, source_t, source_t, bool, spin_t*);
__global__ void create_dilution_mask_k(spin_t*, float*, unsigned int*);
__global__ void fill_lattice_nans_random(spin_t*, spin_t*);
__global__ void fill_lattice_nans_averaged(spin_t*, float*);
__global__ void fill_lattice_nans_averaged_global(spin_t*, spin_t);
__global__ void data_reconstruction_k(source_t*, spin_t*, source_t, source_t, source_t*, source_t*);
__global__ void mean_stdDev_reconstructed_k(source_t*, source_t*, unsigned int);
__global__ void sum_prediction_errors_k(source_t*, source_t*, spin_t*, source_t*, source_t*, source_t*, source_t*, source_t*, source_t*);
__global__ void sum_prediction_errors_k(source_t*, source_t*, spin_t*, source_t*, source_t*, source_t*, source_t*);
__global__ void bondCount_k(spin_t*, unsigned int*);
__global__ void find_temperature_gpu(energy_t*, double*, double*, energy_t*, int, int);
__global__ void correctTemps(energy_t*, energy_t);
__global__ void smoothTemps(energy_t*, energy_t*, energy_t*, bool);
__global__ void setInitialLocalAlphas(spin_t*, spin_t*);
__global__ void setLocalAlphas(unsigned int*, spin_t*, int, energy_t);
std::vector<energy_t> smoothTempsCPU(energy_t*);
energy_t cpu_energy(spin_t*);
double find_temperature(energy_t, std::vector<double>, std::vector<double>);

template <class T> T sumPartialSums(T *, int);
template <class T> std::vector<T> findMinMax(T *, T *, int);
template <class T> T find_median(T *, int);
template <int BLOCK_THREADS, int ITEMS_PER_THREAD> __global__ void BlockSortKernel(energy_t *d_in, energy_t *d_out);


int main()
{

    std::cout << "---Single checkerboard algorithm with local smooth temperatures---\n"

        << "\nRECONSTRUCTION SIMULATION CONFIGURATION:\n"
        << "L = " << L << ",\tQfactor = " << Qfactor << "\n"
        << "BLOCKL = " << BLOCKL << "\n"
        << "Missing data = " << RemovedDataRatio * 100 << "%\n"
        << "Equilibration samples for convergence testing = " << EQUI_TEST_SAMPLES << "\n"
        << "Reconstruction samples = " << SWEEPS_GLOBAL << "\n"
        << "Configuration samples = " << CONFIG_SAMPLES << "\n" << "Active macros: ";
#ifdef DOUBLE_PRECISION
    std::cout << " DOUBLE_PRECISION,";
#else
    std::cout << " SINGLE_PRECISION,";
#ifdef INTRINSIC_FLOAT
    std::cout << " INTRINSIC_FLOAT,";
#endif
#endif
#ifdef ENERGIES_PRINT
    std::cout << " ENERGIES_PRINT,";
#endif
#ifdef CONFIGURATION_PRINT
    std::cout << " CONFIGURATION_PRINT,";
#endif
#ifdef RECONSTRUCTION_PRINT
    std::cout << " RECONSTRUCTION_PRINT,";
#endif
#ifdef ERROR_PRINT
    std::cout << " ERROR_PRINT,";
#endif
#ifdef TEMPSnALPHAS_PRINT
    std::cout << " TEMPSnALPHAS_PRINT,";
#endif 

    
#ifdef OVER_RELAXATION_EQ
    std::cout << " OVER_RELAXATION_EQ,";
#endif
#ifdef OVER_RELAXATION_SIM
    std::cout << " OVER_RELAXATION_SIM,";
#endif
#ifdef SOURCE_MAPPING
    std::cout << " SOURCE_MAPPING,";
#endif
#ifdef RANDOM_INIT
    std::cout << " RANDOM_INIT,";
#endif
    std::cout << "\n";

    // time measurement - entire process
    std::chrono::high_resolution_clock::time_point t_sim_begin = std::chrono::high_resolution_clock::now();

    /* time measurement - relevant parts for geostatistical calulation
    (loading reference E = E(T), loading source, mapping to XY model, equilibration and reconstruction sample collection)
    */
    std::chrono::high_resolution_clock::time_point t_geo_begin;
    std::chrono::high_resolution_clock::time_point t_geo_end;

    t_geo_begin = std::chrono::high_resolution_clock::now();

    //std::cout << "------ LOADING REFERENCES AND SOURCE DATA ------\n";

    // read reference energies and temperatures
    char *buffer;
    const int ref_size = 1100;
    std::vector<double> T_ref;
    std::ifstream fileT("./reference/reference_T.bin", std::ios::in | std::ios::binary);
    buffer = (char*)malloc(ref_size * sizeof(double));
    fileT.read(buffer, ref_size * sizeof(double));
    T_ref.assign(reinterpret_cast<double*>(buffer), reinterpret_cast<double*>(buffer) + ref_size);
    fileT.close();

    std::vector<double> E_ref;
    std::ifstream fileE("./reference/reference_E.bin", std::ios::in | std::ios::binary);
    fileE.read(buffer, ref_size * sizeof(double));
    E_ref.assign(reinterpret_cast<double*>(buffer), reinterpret_cast<double*>(buffer) + ref_size);
    fileE.close();

    free(buffer);

    //allocate memory and copy reference energies and temperatures to the GPU
    double *T_ref_d, *E_ref_d;
    CUDAErrChk(hipMalloc((void**)&T_ref_d, ref_size * sizeof(double)));
    CUDAErrChk(hipMalloc((void**)&E_ref_d, ref_size * sizeof(double)));
    CUDAErrChk(hipMemcpy(T_ref_d, T_ref.data(), ref_size * sizeof(double), hipMemcpyHostToDevice));
    CUDAErrChk(hipMemcpy(E_ref_d, E_ref.data(), ref_size * sizeof(double), hipMemcpyHostToDevice));

    
    // read data source
#ifdef SOURCE_DATA_PATH
    std::cout << "Source data: " << SOURCE_DATA_PATH << "\n";
    std::ifstream fileSource(SOURCE_DATA_PATH, std::ios::in | std::ios::binary);
    std::vector<source_t> complete_source;
    buffer = (char*)malloc(N * sizeof(source_t));
    fileSource.read(buffer, N * sizeof(source_t));
    complete_source.assign(reinterpret_cast<source_t*>(buffer), reinterpret_cast<source_t*>(buffer) + N);
    fileSource.close();
    free(buffer);
#else
    std::cout << "Source data path not specified!";
    return 0;
#endif
    std::cout << "Source size: " << complete_source.size() << "\n";


    //hipSetDevice(0);


    // allocate GPU memory for source data, mapped data (XY model) and dilution mask (array of ones and NANs) & other variables
    source_t *source_d, *reconstructed_d, *mean_recons_d, *stdDev_recons_d, *AAE_d, *ARE_d, *AARE_d, *RASE_d;
#ifdef ERROR_PRINT
    source_t *error_map_d, *error_map_block_d;
    CUDAErrChk(hipMalloc((void**)&error_map_d, N * sizeof(source_t)));
    CUDAErrChk(hipMemset(error_map_d, 0.0, N * sizeof(source_t)));
    CUDAErrChk(hipMalloc((void**)&error_map_block_d, GRIDL * GRIDL * sizeof(source_t)));
    CUDAErrChk(hipMemset(error_map_block_d, 0.0, GRIDL * GRIDL * sizeof(source_t)));
#endif

    spin_t *XY_mapped_d, *dilution_mask_d;
    energy_t *E_d;

    spin_t *alphas_per_block_d, *block_min_d, *block_max_d, *avg_per_block_d, *local_alphas_d;
    energy_t *T_diluted_per_block_d, *localTemps_d, *intermediate_temps_d;
    unsigned int *acc_local_d;
    CUDAErrChk(hipMalloc((void **)&T_diluted_per_block_d, GRIDL * GRIDL * sizeof(energy_t)));
    CUDAErrChk(hipMalloc((void **)&alphas_per_block_d, GRIDL * GRIDL * sizeof(spin_t)));
    CUDAErrChk(hipMalloc((void **)&local_alphas_d, N * sizeof(spin_t)));
    CUDAErrChk(hipMalloc((void **)&block_min_d, GRIDL * GRIDL * sizeof(spin_t)));
    CUDAErrChk(hipMalloc((void **)&block_max_d, GRIDL * GRIDL * sizeof(spin_t)));
    CUDAErrChk(hipMalloc((void **)&avg_per_block_d, GRIDL * GRIDL * sizeof(spin_t)));
    CUDAErrChk(hipMalloc((void **)&localTemps_d, N * sizeof(energy_t)));
    CUDAErrChk(hipMalloc((void **)&intermediate_temps_d, N * sizeof(energy_t)));
    CUDAErrChk(hipMalloc((void **)&acc_local_d, N * sizeof(unsigned int)));
    CUDAErrChk(hipMemset(acc_local_d, 0, N * sizeof(unsigned int)));
    std::vector<spin_t> local_alphas_h(N);
    


    energy_t *AccD;
    unsigned int* tryD;

    CUDAErrChk(hipMalloc((void**)&source_d, N * sizeof(source_t)));
    CUDAErrChk(hipMalloc((void**)&reconstructed_d, N * sizeof(source_t)));
    CUDAErrChk(hipMalloc((void**)&XY_mapped_d, N * sizeof(spin_t)));
    CUDAErrChk(hipMalloc((void**)&dilution_mask_d, N * sizeof(spin_t)));

    CUDAErrChk(hipMalloc((void**)&mean_recons_d, N * sizeof(source_t)));
    CUDAErrChk(hipMalloc((void**)&stdDev_recons_d, N * sizeof(source_t)));

    CUDAErrChk(hipMalloc((void**)&AAE_d, (int)ceil(N / 256.0) * sizeof(source_t)));
    CUDAErrChk(hipMalloc((void**)&ARE_d, (int)ceil(N / 256.0) * sizeof(source_t)));
    CUDAErrChk(hipMalloc((void**)&AARE_d, (int)ceil(N / 256.0) * sizeof(source_t)));
    CUDAErrChk(hipMalloc((void**)&RASE_d, (int)ceil(N / 256.0) * sizeof(source_t)));

    CUDAErrChk(hipMalloc((void **)&E_d, GRIDL * GRIDL * sizeof(energy_t)));

    CUDAErrChk(hipMalloc((void**)&AccD, GRIDL * GRIDL * sizeof(energy_t)));
    CUDAErrChk(hipMalloc((void**)&tryD, GRIDL * GRIDL * sizeof(unsigned int)));

    // for calculating maximum and minimum of data
    source_t *min_d, *max_d;
    CUDAErrChk(hipMalloc((void**)&min_d, (int)ceil(N / 2.0 / 256.0) * sizeof(source_t)));
    CUDAErrChk(hipMalloc((void**)&max_d, (int)ceil(N / 2.0 / 256.0) * sizeof(source_t)));

    std::vector<source_t> min_max;

    // copy source data to GPU memory
    CUDAErrChk(hipMemcpy(source_d, complete_source.data(), N * sizeof(source_t), hipMemcpyHostToDevice));


#ifdef SOURCE_MAPPING
    // ----- MAPPING PROCESS -----
    std::cout << "------ SOURCE MAPPING PROCESS ------\n";

    min_max_k << < gridLinearLatticeHalf, blockLinearLattice >> > (source_d, min_d, max_d, false, dilution_mask_d);
    CUDAErrChk(hipPeekAtLastError());

    min_max = findMinMax(min_d, max_d, (int)ceil(N / 2.0 / 256.0));

    std::cout.precision(17);
    std::cout << "from GPU:  min(source) = " << min_max.at(0)
        << " ; max(source) = " << min_max.at(1) << "\n";
    std::cout.precision(6);

    // mapping to XY model based on max and min
    XY_mapping_k << < gridLinearLattice, blockLinearLattice >> > (source_d, XY_mapped_d, min_max.at(0), min_max.at(1), false, dilution_mask_d);
    CUDAErrChk(hipPeekAtLastError());

    // calculate energy
    energyCalc_k << < gridEn, blockEn >> > (XY_mapped_d, E_d);
    CUDAErrChk(hipPeekAtLastError());
    energy_t E_source = sumPartialSums(E_d, (int)GRIDL * GRIDL) / Nbond;

    // assign temperature
    energy_t T_source = find_temperature(E_source, T_ref, E_ref);
    std::cout << "Source energy per bond: " << E_source << "\n";
    std::cout << "Source temperature: " << T_source << "\n";
#endif

    // print energies
#ifdef ENERGIES_PRINT
    // energies file name + create
    char fileGpuEn[100];
    char fileGpuEnEQ[100];

#ifdef DOUBLE_PRECISION
    sprintf(fileGpuEn, "./data/gpuEn_DP_removed%0.3f_Q%0.3f_L%d_ConfSamples%d_SwGlob%d.dat",
        RemovedDataRatio, (double)Qfactor, L, CONFIG_SAMPLES, SWEEPS_GLOBAL);
    sprintf(fileGpuEnEQ, "./data/gpuEnEQ_DP_removed%0.3f_Q%0.3f_L%d_ConfSamples%d_SwGlob%d.dat",
        RemovedDataRatio, (double)Qfactor, L, CONFIG_SAMPLES, SWEEPS_GLOBAL);
#else
#ifdef INTRINSIC_FLOAT
    sprintf(fileGpuEn, "./data/Energy_SIM_Smooth_p%0.2f_M%d_%s.dat",
        RemovedDataRatio, CONFIG_SAMPLES, SOURCE_DATA_NAME);
    sprintf(fileGpuEnEQ, "./data/Energy_EQ_Smooth_p%0.2f_M%d_%s.dat",
        RemovedDataRatio, CONFIG_SAMPLES, SOURCE_DATA_NAME);
#else
    sprintf(fileGpuEn, "./data/gpuEn_SP_removed%0.3f_Q%0.2f_L%d_ConfSamples%d_SwGlob%d.dat",
        RemovedDataRatio, (double)Qfactor, L, CONFIG_SAMPLES, SWEEPS_GLOBAL);
    sprintf(fileGpuEnEQ, "./data/gpuEnEQ_removed%0.3f_SP_Q%0.2f_L%d_ConfSamples%d_SwGlob%d.dat",
        RemovedDataRatio, (double)Qfactor, L, CONFIG_SAMPLES, SWEEPS_GLOBAL);
#endif
#endif

    FILE *fp = fopen(fileGpuEn, "wb");
    FILE *fpEQ = fopen(fileGpuEnEQ, "wb");
#endif

    // store output data
#ifdef RECONSTRUCTION_PRINT
    char fileMean[100];
    //char fileStdDev[100];

    sprintf(fileMean, "./data/Recons_Smooth_%d_p%0.2f_M%d_%s.dat",
        SMOOTHING_PARAMETER, RemovedDataRatio, CONFIG_SAMPLES, SOURCE_DATA_NAME);
        
    FILE *fpMean = fopen(fileMean, "wb");
    //FILE *fpStdDev = fopen(fileStdDev, "wb");
#endif

#ifdef ERROR_PRINT
    char fileError[100];
    char fileErrorBlock[100];
    sprintf(fileError, "./data/Error_Smooth_p%0.2f_M%d_%s.dat",
        RemovedDataRatio, CONFIG_SAMPLES, SOURCE_DATA_NAME);
    sprintf(fileErrorBlock, "./data/Error_Smooth_Block_p%0.2f_M%d_%s.dat",
        RemovedDataRatio, CONFIG_SAMPLES, SOURCE_DATA_NAME);
    FILE *fpError = fopen(fileError, "wb");
    FILE *fpErrorBlock = fopen(fileErrorBlock, "wb");
#endif

#ifdef TEMPSnALPHAS_PRINT
    char fileTemps_block[100];
    char fileTemps_local[100];
    char fileAlphas_local_eq[100];
    char fileAlphas_local_sim[100];
    sprintf(fileTemps_block, "./data/Temps_Block_Smooth_p%0.2f_M%d_%s.dat",
        RemovedDataRatio, CONFIG_SAMPLES, SOURCE_DATA_NAME);
    sprintf(fileTemps_local, "./data/Temps_Local_Smooth_%d_p%0.2f_M%d_%s.dat",
        SMOOTHING_PARAMETER, RemovedDataRatio, CONFIG_SAMPLES, SOURCE_DATA_NAME);
    sprintf(fileAlphas_local_eq, "./data/Alphas_Local_EQ_Smooth_p%0.2f_M%d_%s.dat",
        RemovedDataRatio, CONFIG_SAMPLES, SOURCE_DATA_NAME);
    sprintf(fileAlphas_local_sim, "./data/Alphas_Local_SIM_Smooth_p%0.2f_M%d_%s.dat",
        RemovedDataRatio, CONFIG_SAMPLES, SOURCE_DATA_NAME);
    FILE *fpTempsBlock = fopen(fileTemps_block, "wb");
    FILE *fpTempsLocal = fopen(fileTemps_local, "wb");
    FILE *fpAlphasLocal_Eq = fopen(fileAlphas_local_eq, "wb");
    FILE *fpAlphasLocal_SIM = fopen(fileAlphas_local_sim, "wb");
#endif

#ifdef CONFIGURATION_PRINT
    // print diluted data into file
    spin_t *mask;
    mask = (spin_t*)malloc(N * sizeof(spin_t));
    CUDAErrChk(hipMemcpy(mask, dilution_mask_d, N * sizeof(spin_t), hipMemcpyDeviceToHost));
    char strConf[100];
    sprintf(strConf, "./data/conf_removed%1.3f_Q%0.3f_L%d_ConfSamples%d_SwGlob%d.dat",
        RemovedDataRatio, (double)Qfactor, L, CONFIG_SAMPLES, SWEEPS_GLOBAL);

    FILE *f_conf = fopen(strConf, "wb");
#endif

    // SEEDS
    unsigned long long seed_dilution;
    unsigned long long seed_fill;
    unsigned long long seed_simulation;

    // calculation of configurational means
    source_t MAAE = 0.0, MARE = 0.0, MAARE = 0.0, MRASE = 0.0,
        M_timeEQ = 0.0, M_timeSamples = 0.0;
    int sum_eqSw = 0;

    t_geo_end = std::chrono::high_resolution_clock::now();
    long long duration_initial = std::chrono::duration_cast<std::chrono::microseconds>(t_geo_end - t_geo_begin).count();
    long long duration_mapping_EQ_sampling = 0;
    

    /*
    --------------------------------------------------------------
    --------------- LOOP FOR CONFIGURATION SAMPLES ---------------
    --------------------------------------------------------------
    */
    for (int n = 0; n < CONFIG_SAMPLES; ++n)
    {
        // ----- GPU DILUTION ------
        //std::cout << "------ GPU DILUTION ------\n";
        // creating RN generator for dilution
        float *devRand_dil;
        unsigned int *remSum_d;
        CUDAErrChk(hipMalloc((void **)&devRand_dil, N * sizeof(float)));
        CUDAErrChk(hipMalloc((void **)&remSum_d, (int)ceil(N / 256.0) * sizeof(unsigned int)));

        hiprandGenerator_t RNgen_dil;
        hiprandStatus_t err; // hiprand errors
        err = hiprandCreateGenerator(&RNgen_dil, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        cuRAND_ErrChk(err);

        // setting seed
        seed_dilution = (n == 0) ?
#ifdef RNG_SEED_DILUTION 
            RNG_SEED_DILUTION
#else
            time(NULL)
#endif
            : RAN(seed_dilution);

        err = hiprandSetPseudoRandomGeneratorSeed(RNgen_dil, seed_dilution);
        cuRAND_ErrChk(err);
        // generate random floats on device - for every spin in the lattice and for every local sweep
        err = hiprandGenerateUniform(RNgen_dil, devRand_dil, N);
        cuRAND_ErrChk(err);


        create_dilution_mask_k << < gridLinearLattice, blockLinearLattice >> > (dilution_mask_d, devRand_dil, remSum_d);
        CUDAErrChk(hipPeekAtLastError());
        int removedTotal = sumPartialSums(remSum_d, (int)ceil(N / 256.0));

        //std::cout << "Removed = " << removedTotal << " , Removed data ratio = " << removedTotal / (double)N << "\n";

        // RNG cease activity here
        hiprandDestroyGenerator(RNgen_dil);
        CUDAErrChk(hipFree(devRand_dil));
        CUDAErrChk(hipFree(remSum_d));

        // time measurement - relevant part for geostatistical application
        t_geo_begin = std::chrono::high_resolution_clock::now();

        // calculate number of bonds in diluted system
        unsigned int *bondCount_d;

        CUDAErrChk(hipMalloc((void **)&bondCount_d, GRIDL * GRIDL * sizeof(unsigned int)));
        bondCount_k << < gridEn, blockEn >> > (dilution_mask_d, bondCount_d);
        CUDAErrChk(hipPeekAtLastError());
        int Nbonds_dil = sumPartialSums(bondCount_d, GRIDL * GRIDL);

        // mapping diluted system to XY model
        min_max_k << < gridLinearLatticeHalf, blockLinearLattice >> > (source_d, min_d, max_d, true, dilution_mask_d);
        CUDAErrChk(hipPeekAtLastError());

        min_max = findMinMax(min_d, max_d, (int)ceil(N / 2.0 / 256.0));

        /*
        std::cout.precision(17);
        std::cout << "from GPU:  min(diluted) = " << min_max.at(0)
        << " ; max(diluted) = " << min_max.at(1) << "\n";
        std::cout.precision(6);
        */

        // mapping to XY model based on max and min
        XY_mapping_k << < gridLinearLattice, blockLinearLattice >> > (source_d, XY_mapped_d, min_max.at(0), min_max.at(1), true, dilution_mask_d);
        CUDAErrChk(hipPeekAtLastError());

        // calculate energy
        energyCalcDiluted_k << < gridEn, blockEn >> > (XY_mapped_d, E_d);
        CUDAErrChk(hipPeekAtLastError());
        energy_t E_diluted = sumPartialSums(E_d, (int)GRIDL * GRIDL) / (energy_t)Nbonds_dil;
        // assign a single temperature
        energy_t T_avg = find_temperature(E_diluted, T_ref, E_ref);
        //std::cout << "Diluted - energy per bond: " << E_diluted << "\n";
        //std::cout << "Diluted - temperature: " << T_diluted << "\n";
        //calculate the energy per bond separately in each block

        //number of per-block results
        const int size = GRIDL * GRIDL;
        int n_blocks = std::min((size + 1024 - 1) / 1024, 1024);
        energyCalcDiluted_per_block << <n_blocks, 1024 >> > (E_d, bondCount_d, size);
       // std::cout << "avg bonds per block = " << (energy_t)Nbonds_dil / size << "\n";
        CUDAErrChk(hipFree(bondCount_d));

        min_max_avg_block << < gridEn, blockEn >> > (XY_mapped_d, block_min_d, block_max_d, avg_per_block_d);
        // assign a different temperature to each block
        find_temperature_gpu << <n_blocks, 1024 >> > (E_d, T_ref_d, E_ref_d, T_diluted_per_block_d, size, ref_size);
        energy_t T_diluted, T_median;
        T_median = find_median(T_diluted_per_block_d, size);
        /*
        energy_t* d_sorted_temps;
        CUDAErrChk(hipMalloc((void **)&d_sorted_temps, size * sizeof(energy_t)));
        const int num_threads = std::min(size, 1024);
        const int items_per_thread = std::max(size / 1024, 1);
        //sorting block temperatures for finding median block temperature and minimum block temperature
        BlockSortKernel<num_threads, items_per_thread> << <1, num_threads >> >(T_diluted_per_block_d, d_sorted_temps);
        energy_t value1, value2, minT;
        hipMemcpy(&value1, &d_sorted_temps[(size - 1) / 2], sizeof(energy_t), hipMemcpyDeviceToHost);
        hipMemcpy(&value2, &d_sorted_temps[size / 2], sizeof(energy_t), hipMemcpyDeviceToHost);
        T_median = (value1 + value2) / 2;
        //correcting block temperatures in case there were no bond in some blocks, these blocks get the median temperature instead
        correctTemps << <n_blocks, 1024 >> > (T_diluted_per_block_d, T_median);
        //sorting again after correction
        BlockSortKernel<num_threads, items_per_thread> << <1, num_threads >> >(T_diluted_per_block_d, d_sorted_temps);
        hipMemcpy(&value1, &d_sorted_temps[(size - 1) / 2], sizeof(energy_t), hipMemcpyDeviceToHost);
        hipMemcpy(&value2, &d_sorted_temps[size / 2], sizeof(energy_t), hipMemcpyDeviceToHost);
        T_median = (value1 + value2) / 2;
        hipMemcpy(&minT, &d_sorted_temps[0], sizeof(energy_t), hipMemcpyDeviceToHost);
        */
        T_diluted = T_median;

        //CUDAErrChk(hipFree(d_sorted_temps));
        //smoothing block temperatures, creating local temperatures
        smoothTemps << <gridEn, blockEn >> > (T_diluted_per_block_d, intermediate_temps_d, localTemps_d, true);

        for (unsigned int i = 0; i < SMOOTHING_PARAMETER; i++)
        {
            smoothTemps << <gridEn, blockEn >> > (T_diluted_per_block_d, intermediate_temps_d, localTemps_d, false);
            CUDAErrChk(hipMemcpy(intermediate_temps_d, localTemps_d, N * sizeof(energy_t), hipMemcpyDeviceToDevice));
            
        }


        if (n == 0)
        {
            std::cout << "Diluted - energy per bond: " << E_diluted << "\n";
            std::cout << "Diluted - temperature: " << T_avg << "\n";
            //std::cout << "Median temperature = " << T_median << "\n";
            //std::cout << "minimum temperature = " << minT << "\n";
            /*
            energy_t* h_sorted_temps;
            hipMemcpy(h_sorted_temps, d_sorted_temps, size * sizeof(energy_t), hipMemcpyDeviceToHost);
            CUDAErrChk(hipFree(d_sorted_temps));
            std::cout << "median temperature CPU = " << T_diluted << " median temperature GPU = " << (value1 + value2) / 2 << "\n\n";
            //std::cout << " median temperature GPU = " << (value1 + value2) / 2 << "\n";

            for (int i = 0; i < size; i++)
            {
            std::cout << h_sorted_temps[i] << "\t";
            }
            std::cout << "\n\n";
            */

#ifdef TEMPSnALPHAS_PRINT
            std::vector<energy_t> blockTemps_h;
            std::vector<energy_t> localTemps_h;
            blockTemps_h.resize(GRIDL * GRIDL);
            localTemps_h.resize(L * L);
            CUDAErrChk(hipMemcpy(blockTemps_h.data(), T_diluted_per_block_d, sizeof(energy_t) * GRIDL * GRIDL, hipMemcpyDeviceToHost));
            CUDAErrChk(hipMemcpy(localTemps_h.data(), localTemps_d, sizeof(energy_t) * L * L, hipMemcpyDeviceToHost));

            for (int k = 0; k < N; ++k)
                fwrite(&(localTemps_h.at(k)), sizeof(energy_t), 1, fpTempsLocal);

            
            for (int k = 0; k < GRIDL * GRIDL; k++)
            {
                fwrite(&(blockTemps_h.at(k)), sizeof(energy_t), 1, fpTempsBlock);         
            }
            //std::cout << "END OF ITERATION " << n << "\n\n";
                        
#endif
        }
        
#ifdef CONFIGURATION_PRINT
            // print diluted data into file
            spin_t *mask;
            mask = (spin_t*)malloc(N * sizeof(spin_t));
            CUDAErrChk(hipMemcpy(mask, dilution_mask_d, N * sizeof(spin_t), hipMemcpyDeviceToHost));

            for (int i = 0; i < N; ++i)
            {
                source_t temp = complete_source.at(i) * mask[i];
                fwrite(&temp, sizeof(source_t), 1, f_conf);
            }
            //std::cout << "I'm here after writing" << "\n\n";
#endif
        
        // ------ FILLING NAN VALUES WITH RANDOM SPINS ------
        //std::cout << "------ FILLING NAN VALUES WITH RANDOM SPINS ------\n";
        // creating RN generator for dilution
                
#ifdef RANDOM_INIT
        float *devRand_fill;
        CUDAErrChk(hipMalloc((void **)&devRand_fill, N * sizeof(float)));

        hiprandGenerator_t RNgen_fill;
        err = hiprandCreateGenerator(&RNgen_fill, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        cuRAND_ErrChk(err);

        // setting seed
        seed_fill = (n == 0) ?
#ifdef RNG_SEED_FILL 
            RNG_SEED_FILL
#else
            time(NULL)
#endif
            : RAN(seed_fill);

        err = hiprandSetPseudoRandomGeneratorSeed(RNgen_fill, seed_fill);
        cuRAND_ErrChk(err);
        // generate random floats on device - for every spin site
        err = hiprandGenerateUniform(RNgen_fill, devRand_fill, N);
        cuRAND_ErrChk(err);
        fill_lattice_nans_random << < gridLinearLattice, blockLinearLattice >> > (XY_mapped_d, devRand_fill);
        // RNG cease activity here
        hiprandDestroyGenerator(RNgen_fill);
        CUDAErrChk(hipFree(devRand_fill));
#else
        //spin_t global_average = sumPartialSums(avg_per_block_d, (int)GRIDL * GRIDL) / (GRIDL * GRIDL);
        //fill_lattice_nans_averaged_global << < gridEn, blockEn >> > (XY_mapped_d, global_average);
        fill_lattice_nans_averaged << < gridEn, blockEn >> > (XY_mapped_d, avg_per_block_d);
#endif 
        CUDAErrChk(hipPeekAtLastError());          

        // ------ CONDITIONED MC SIMULATION -----
        //std::cout << "------ GPU CONDITIONED MC SIMULATION ------\n";
        // create data arrays for thermodynamic variables
        std::vector<energy_t> EnergiesEq;
        std::vector<energy_t> Energies(SWEEPS_GLOBAL);

        // creating RN generator for equilibration and simulation
        // setting seed
        seed_simulation = (n == 0) ?
#ifdef RNG_SEED_SIMULATION 
            RNG_SEED_SIMULATION
#else
            time(NULL)
#endif
            : RAN(seed_simulation);

        // creating RN generator for equilibration and simulation
        float* devRand;
        energy_t alpha = (energy_t)2 * M_PI;

        //std::cout << "iteracia " << n <<"\n";
        
        CUDAErrChk(hipMalloc((void **)&devRand, 2 * N * SWEEPS_EMPTY * sizeof(float)));

        hiprandGenerator_t RNgen;
        err = hiprandCreateGenerator(&RNgen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        cuRAND_ErrChk(err);
        err = hiprandSetPseudoRandomGeneratorSeed(RNgen, seed_simulation);
        cuRAND_ErrChk(err);

        // summation of reconstructed data for means and standard deviations
        std::vector<source_t> mean_reconstructed(N, 0.0);
        std::vector<source_t> stdDev_reconstructed(N, 0.0);
        CUDAErrChk(hipMemcpy(mean_recons_d, mean_reconstructed.data(), N * sizeof(source_t), hipMemcpyHostToDevice));
        CUDAErrChk(hipMemcpy(stdDev_recons_d, stdDev_reconstructed.data(), N * sizeof(source_t), hipMemcpyHostToDevice));

        // event creation
        hipEvent_t start, stop, startEq, stopEq;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventCreate(&startEq);
        hipEventCreate(&stopEq);
        float Etime;
        float EtimeEq;

        // start measurment

        hipEventRecord(startEq, 0);

        // ------ EQUILIBRATION ------
        //std::cout << "------ EQUILIBRATION ------\n";
        // acceptance rate + adjustment of spin-perturbation interval parameter "alpha"
        //float alpha = (float)(2.0*M_PI);
        //double AccRate;

        std::vector<energy_t> AccH(GRIDL * GRIDL, 0.0);
        std::vector<unsigned int> tryH(GRIDL * GRIDL, 0);
        setInitialLocalAlphas << < gridLinearLattice, blockLinearLattice >> > (local_alphas_d, dilution_mask_d);
        //reset accepted flips
        hipMemset(acc_local_d, 0, N * sizeof(unsigned int));
        //std::vector<energy_t> alphas_H(GRIDL * GRIDL, (energy_t)(2.0*M_PI));
        CUDAErrChk(hipMemcpy(AccD, AccH.data(), GRIDL * GRIDL * sizeof(energy_t), hipMemcpyHostToDevice));
        CUDAErrChk(hipMemcpy(tryD, tryH.data(), GRIDL * GRIDL * sizeof(unsigned int), hipMemcpyHostToDevice));
        //CUDAErrChk(hipMemcpy(alphas_per_block_d, alphas_H.data(), GRIDL * GRIDL * sizeof(energy_t), hipMemcpyHostToDevice));

#ifdef ENERGIES_PRINT
        // Calculate initial energy and write it into file
        energyCalc_k << < gridEn, blockEn >> > (XY_mapped_d, E_d);
        CUDAErrChk(hipPeekAtLastError());
        energy_t energy = sumPartialSums(E_d, (int)GRIDL * GRIDL) / Nbond;
        //EnergiesEq.push_back(energy);

        fwrite(&(energy), sizeof(energy_t), 1, fpEQ);
#endif


        // slope of simple linear regression
        energy_t Slope = -1;
        int it_EQ = 1;
        energy_t meanX = EQUI_TEST_SAMPLES / (energy_t)2.0;
        energy_t varX = 0.0;
        std::vector<energy_t> Xdiff;
        for (int i = 0; i < EQUI_TEST_SAMPLES; ++i)
        {
            Xdiff.push_back(i - meanX);
            varX += Xdiff.at(i) * Xdiff.at(i);
            //std::cout << "Xdiff[i] =  " << Xdiff[i] << "\n";
        }

        Slope = -1;

        while ((Slope < 0) && (it_EQ <= SWEEPS_EQUI_MAX))
        {
            
#ifdef OVER_RELAXATION_EQ
            // over-relaxation algorithm
            spin_mult << < gridLinearLattice, blockLinearLattice >> > (XY_mapped_d, Qfactor);
            CUDAErrChk(hipPeekAtLastError());
            over_relaxation_k << < grid_check, block_check >> > (XY_mapped_d, dilution_mask_d, 0);
            CUDAErrChk(hipPeekAtLastError());
            over_relaxation_k << < grid_check, block_check >> > (XY_mapped_d, dilution_mask_d, 1);
            CUDAErrChk(hipPeekAtLastError());
            spin_mult << < gridLinearLattice, blockLinearLattice >> > (XY_mapped_d, 1 / Qfactor);
            CUDAErrChk(hipPeekAtLastError());
#endif

            err = hiprandGenerateUniform(RNgen, devRand, 2 * N * SWEEPS_EMPTY);
            cuRAND_ErrChk(err);
            for (int j = 0; j < SWEEPS_EMPTY; ++j)
            {
                metro_conditioned_sublattice_k << < grid_check, block_check >> > (XY_mapped_d, dilution_mask_d, devRand + j*N, 0, localTemps_d, acc_local_d, local_alphas_d);
                CUDAErrChk(hipPeekAtLastError());
                metro_conditioned_sublattice_k << < grid_check, block_check >> > (XY_mapped_d, dilution_mask_d, devRand + j*N, 1, localTemps_d, acc_local_d, local_alphas_d);
                CUDAErrChk(hipPeekAtLastError());
            }
            //std::cout << "it_EQ = " << it_EQ << "\n";
            setLocalAlphas << < gridEn, blockEn >> > (acc_local_d, local_alphas_d, it_EQ, ACC_RATE_MIN_EQ);
            CUDAErrChk(hipPeekAtLastError());
            // energy calculation and sample filling
            energyCalc_k << < gridEn, blockEn >> > (XY_mapped_d, E_d);
            CUDAErrChk(hipPeekAtLastError());
            energy_t energy = sumPartialSums(E_d, (int)GRIDL * GRIDL) / Nbond;
            EnergiesEq.push_back(energy);
#ifdef ENERGIES_PRINT
            fwrite(&(EnergiesEq.back()), sizeof(energy_t), 1, fpEQ);
#endif

            // keeps the number of energy samples stable (= EQUI_TEST_SAMPLES)
            if (EnergiesEq.size() > EQUI_TEST_SAMPLES)
                EnergiesEq.erase(EnergiesEq.begin());

            ++it_EQ;	// iterator update ("it_EQ = 1" for 1st hybrid sweep)
                        //std::cout << "iteration_global number " << it_EQ << " has energy " << energy <<"\n";
                        // Slope update
            if ((it_EQ % EQUI_TEST_FREQUENCY) == 0)
            {
                // testing equilibration condition - claculation of linear regression slope from stored energies
                if (EnergiesEq.size() == EQUI_TEST_SAMPLES)
                {
                    energy_t sumEn = 0.0;
                    for (auto n : EnergiesEq) sumEn += n;
                    energy_t meanEn = sumEn / EQUI_TEST_SAMPLES;
                    sumEn = 0.0;
                    for (int k = 0; k < EQUI_TEST_SAMPLES; ++k)
                        sumEn += (EnergiesEq.at(k) - meanEn) * Xdiff.at(k);
                    Slope = sumEn / varX;
                    //std::cout << "Slope_global =  " << Slope << "\n";
                }
            }
        }

        // end measurment
        CUDAErrChk(hipEventRecord(stopEq, 0));
        CUDAErrChk(hipEventSynchronize(stopEq));
        CUDAErrChk(hipEventElapsedTime(&EtimeEq, startEq, stopEq));

#ifdef ENERGIES_PRINT
        for (int i = it_EQ; i <= SWEEPS_EQUI_MAX; i++)
        {
            int k = 0;
            fwrite(&k, sizeof(int), 1, fpEQ);
        }
#endif

#ifdef TEMPSnALPHAS_PRINT
        if (n == 0)
        {
            local_alphas_h.resize(N);
            CUDAErrChk(hipMemcpy(local_alphas_h.data(), local_alphas_d, sizeof(spin_t) * N, hipMemcpyDeviceToHost));

            for (int k = 0; k < N; ++k)
                fwrite(&(local_alphas_h.at(k)), sizeof(spin_t), 1, fpAlphasLocal_Eq);
            //std::cout << "lokalna alfa 4 = " << local_alphas_h[4] << "\n";
        }
#endif 
        // start measurment

        hipEventRecord(start, 0);

        // ------ GENERATING SAMPLES ------
        //single checkerboard version
        for (int i = 0; i < SWEEPS_GLOBAL; ++i)
        {
            
#ifdef OVER_RELAXATION_SIM
            // over-relaxation algorithm
            spin_mult << < gridLinearLattice, blockLinearLattice >> > (XY_mapped_d, Qfactor);
            CUDAErrChk(hipPeekAtLastError());
            over_relaxation_k << < grid_check, block_check >> > (XY_mapped_d, dilution_mask_d, 0);
            CUDAErrChk(hipPeekAtLastError());
            over_relaxation_k << < grid_check, block_check >> > (XY_mapped_d, dilution_mask_d, 1);
            CUDAErrChk(hipPeekAtLastError());
            spin_mult << < gridLinearLattice, blockLinearLattice >> > (XY_mapped_d, 1 / Qfactor);
            CUDAErrChk(hipPeekAtLastError());
#endif
            // generate random floats on device - for every spin in the lattice and for every empty sweep
            err = hiprandGenerateUniform(RNgen, devRand, 2 * N * SWEEPS_EMPTY);
            cuRAND_ErrChk(err);

            for (int j = 0; j < SWEEPS_EMPTY; ++j)
            {
                metro_conditioned_sublattice_k << < grid_check, block_check >> > (XY_mapped_d, dilution_mask_d, devRand + j*N, 0, localTemps_d, acc_local_d, local_alphas_d);
                CUDAErrChk(hipPeekAtLastError());
                metro_conditioned_sublattice_k << < grid_check, block_check >> > (XY_mapped_d, dilution_mask_d, devRand + j*N, 1, localTemps_d, acc_local_d, local_alphas_d);
                CUDAErrChk(hipPeekAtLastError());
            }
            setLocalAlphas << < gridEn, blockEn >> > (acc_local_d, local_alphas_d, it_EQ + i, ACC_RATE_MIN_EQ); 
#ifdef ENERGIES_PRINT
            // energy calculation
            energyCalc_k << < gridEn, blockEn >> > (XY_mapped_d, E_d);
            CUDAErrChk(hipPeekAtLastError());
            Energies.at(i) = sumPartialSums(E_d, (int)GRIDL * GRIDL) / Nbond;
            fwrite(&(Energies.at(i)), sizeof(energy_t), 1, fp);
#endif

            // data reconstruction + summation for mean and standard deviation
            data_reconstruction_k << < gridLinearLattice, blockLinearLattice >> > (reconstructed_d, XY_mapped_d, min_max.at(0), min_max.at(1), mean_recons_d, stdDev_recons_d);
            CUDAErrChk(hipPeekAtLastError());
        }

        // end measurment
        CUDAErrChk(hipEventRecord(stop, 0));
        CUDAErrChk(hipEventSynchronize(stop));
        CUDAErrChk(hipEventElapsedTime(&Etime, start, stop));

        // GPU time
        M_timeEQ += EtimeEq / 1000;
        M_timeSamples += Etime / 1000;

        // prediction averages and standard deviations
        mean_stdDev_reconstructed_k << < gridLinearLattice, blockLinearLattice >> > (mean_recons_d, stdDev_recons_d, SWEEPS_GLOBAL);
        CUDAErrChk(hipPeekAtLastError());

        t_geo_end = std::chrono::high_resolution_clock::now();
        duration_mapping_EQ_sampling += std::chrono::duration_cast<std::chrono::microseconds>(t_geo_end - t_geo_begin).count();

#ifdef RECONSTRUCTION_PRINT
        CUDAErrChk(hipMemcpy(mean_reconstructed.data(), mean_recons_d, N * sizeof(source_t), hipMemcpyDeviceToHost));
        if (n == 0)
        {
            for (int k = 0; k < N; ++k)
            {
                //std::cout << "zapisujem " << mean_reconstructed.at(k) << " a " << stdDev_reconstructed.at(k) << "\n";
                fwrite(&(mean_reconstructed.at(k)), sizeof(source_t), 1, fpMean);
                //fwrite(&(stdDev_reconstructed.at(k)), sizeof(source_t), 1, fpStdDev);
            }
        }
#endif
       
        // does not work with BlockL 8
#ifdef ERROR_PRINT
        // prediction errors
        sum_prediction_errors_k << < gridEn, blockEn >> > (source_d, mean_recons_d, dilution_mask_d, AAE_d, ARE_d, AARE_d, RASE_d, error_map_d, error_map_block_d);
        CUDAErrChk(hipPeekAtLastError());
        MAAE += sumPartialSums(AAE_d, (int)GRIDL * GRIDL) / (source_t)removedTotal;
        MARE += sumPartialSums(ARE_d, (int)GRIDL * GRIDL) / (source_t)removedTotal;
        MAARE += sumPartialSums(AARE_d, (int)GRIDL * GRIDL) / (source_t)removedTotal;
        MRASE += sqrt(sumPartialSums(RASE_d, (int)GRIDL * GRIDL) / (source_t)removedTotal);
#else
        // works with BlockL 8
        // prediction errors
        sum_prediction_errors_k << < gridLinearLattice, blockLinearLattice >> > (source_d, mean_recons_d, dilution_mask_d, AAE_d, ARE_d, AARE_d, RASE_d);
        CUDAErrChk(hipPeekAtLastError());
        MAAE += sumPartialSums(AAE_d, (int)ceil(N / 256.0)) / (source_t)removedTotal;
        MARE += sumPartialSums(ARE_d, (int)ceil(N / 256.0)) / (source_t)removedTotal;
        MAARE += sumPartialSums(AARE_d, (int)ceil(N / 256.0)) / (source_t)removedTotal;
        MRASE += sqrt(sumPartialSums(RASE_d, (int)ceil(N / 256.0)) / (source_t)removedTotal);
#endif
        

        // Number of equilibration sweeps
        sum_eqSw += it_EQ;

        // hipFree after equilibration
        hiprandDestroyGenerator(RNgen);
        CUDAErrChk(hipFree(devRand));
        

        if (n == 0) std::cout << "Seeds[configurations, filling, simulation] = " << "["
            << seed_dilution << ", " << seed_fill << ", " << seed_simulation << "]\n";

    }

    std::cout.precision(8);
    std::cout << "Mean elapsed time (equilibration for average " << sum_eqSw / (source_t)CONFIG_SAMPLES << " sweeps) = " << M_timeEQ / CONFIG_SAMPLES << " s\n";
    std::cout << "Mean elapsed time (collection of " << CONFIG_SAMPLES << " samples) = " << M_timeSamples / CONFIG_SAMPLES << " s\n";

    // prediction errors
    std::cout << "MAAE = " << MAAE / CONFIG_SAMPLES << "\n";
    std::cout << "MARE = " << MARE * 100 / CONFIG_SAMPLES << " %\n";
    std::cout << "MAARE = " << MAARE * 100 / CONFIG_SAMPLES << " %\n";
    std::cout << "MRASE = " << MRASE / CONFIG_SAMPLES << "\n";


#ifdef ERROR_PRINT
    std::vector<source_t> error_map_h(N);
    std::vector<source_t> error_map_block_h(GRIDL * GRIDL);
    CUDAErrChk(hipMemcpy(error_map_h.data(), error_map_d, N * sizeof(source_t), hipMemcpyDeviceToHost));
    CUDAErrChk(hipMemcpy(error_map_block_h.data(), error_map_block_d, GRIDL * GRIDL * sizeof(source_t), hipMemcpyDeviceToHost));

    for (int k = 0; k < N; ++k)
    {
        error_map_h[k] = error_map_h[k] / (source_t)CONFIG_SAMPLES;
        fwrite(&(error_map_h.at(k)), sizeof(source_t), 1, fpError);
    }
    
    for (int k = 0; k < GRIDL * GRIDL; k++)
    {
        error_map_block_h[k] = error_map_block_h[k] / (source_t)CONFIG_SAMPLES;
        fwrite(&(error_map_block_h.at(k)), sizeof(source_t), 1, fpErrorBlock);
    }
#endif

    // closing time series storage
#ifdef ENERGIES_PRINT  
    fclose(fp);
    fclose(fpEQ);
#endif
#ifdef RECONSTRUCTION_PRINT
    fclose(fpMean);
    //fclose(fpStdDev);
#endif
#ifdef CONFIGURATION_PRINT
    fclose(f_conf);
#endif
#ifdef ERROR_PRINT
    CUDAErrChk(hipFree(error_map_d));
    fclose(fpError);
    CUDAErrChk(hipFree(error_map_block_d));
    fclose(fpErrorBlock);
#endif
#ifdef TEMPSnALPHAS_PRINT
    CUDAErrChk(hipMemcpy(local_alphas_h.data(), local_alphas_d, sizeof(spin_t) * N, hipMemcpyDeviceToHost));

    for (int k = 0; k < N; ++k)
        fwrite(&(local_alphas_h.at(k)), sizeof(spin_t), 1, fpAlphasLocal_SIM);
    
    fclose(fpTempsBlock);
    fclose(fpTempsLocal);
    fclose(fpAlphasLocal_Eq);
    fclose(fpAlphasLocal_SIM);
#endif 

    // free CUDA variable
    CUDAErrChk(hipFree(source_d));
    CUDAErrChk(hipFree(XY_mapped_d));
    CUDAErrChk(hipFree(dilution_mask_d));
    CUDAErrChk(hipFree(reconstructed_d));
    CUDAErrChk(hipFree(min_d));
    CUDAErrChk(hipFree(max_d));
    CUDAErrChk(hipFree(E_d));


    CUDAErrChk(hipFree(T_diluted_per_block_d));
    CUDAErrChk(hipFree(alphas_per_block_d));
    CUDAErrChk(hipFree(T_ref_d));
    CUDAErrChk(hipFree(E_ref_d));
    CUDAErrChk(hipFree(tryD));
    CUDAErrChk(hipFree(block_min_d));
    CUDAErrChk(hipFree(block_max_d));
    CUDAErrChk(hipFree(avg_per_block_d));
    CUDAErrChk(hipFree(localTemps_d));
    CUDAErrChk(hipFree(intermediate_temps_d));
    CUDAErrChk(hipFree(local_alphas_d));
    CUDAErrChk(hipFree(acc_local_d));


    CUDAErrChk(hipFree(AccD));
    CUDAErrChk(hipFree(mean_recons_d));
    CUDAErrChk(hipFree(stdDev_recons_d));
    CUDAErrChk(hipFree(AAE_d));
    CUDAErrChk(hipFree(ARE_d));
    CUDAErrChk(hipFree(AARE_d));
    CUDAErrChk(hipFree(RASE_d));

    // time measurement - entire process
    std::chrono::high_resolution_clock::time_point t_sim_end = std::chrono::high_resolution_clock::now();
    auto tot_duration = std::chrono::duration_cast<std::chrono::microseconds>(t_sim_end - t_sim_begin).count();
    std::cout << "Total duration = " << (double)tot_duration / 1e6 << " s\n";
    std::cout << "Total duration per configuration sample = " << (double)tot_duration / 1e6 / CONFIG_SAMPLES << " s\n";
    // time measurement - relevant part for geostatistical application
    //(loading reference E = E(T), loading source, mapping to XY model, equilibration and reconstruction sample collection)
    std::cout << "------DURATION OF GEOSTATISTICAL APPLICATION------\n"
        //<< "Inicialization processes (loading reference E=E(T), loading source data, GPU memory allocation and copying):\n"
        << "t_initialization = " << (double)duration_initial / 1e6 << " s\n"
        //<< "Mapping to XY model, equilibration and reconstruction sample collection (per configuration sample):\n"
        << "t_reconstruction = " << (double)duration_mapping_EQ_sampling / 1e6 / CONFIG_SAMPLES << " s\n"
        //<< "Mapping to XY model, equilibration and reconstruction sample collection:\n"
        << "t_TOTAL = " << ((double)duration_initial / 1e6 + (double)duration_mapping_EQ_sampling / 1e6 / CONFIG_SAMPLES) << " s\n";

    return 0;
}

//single checkerboard versions
__global__ void metro_conditioned_sublattice_k(spin_t *s, spin_t *dilution_mask_d, float *devRand, unsigned int offset, energy_t* localTemps, unsigned int *AccD, spin_t* local_alphas_d)
{
    // int t = threadIdx.x + BLOCKL*threadIdx.y;
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = 2 * threadIdx.y + (threadIdx.x + offset) % 2 + BLOCKL*blockIdx.y;

    unsigned int n = threadIdx.x + threadIdx.y*BLOCKL;
    unsigned int idx = n + THREADS * (blockIdx.x + gridDim.x*blockIdx.y);
    
    if (isnan(dilution_mask_d[x + L*y]))
    {
        energy_t BETA = 1.0 / localTemps[x + L*y];
        spin_t S_old = s[x + L*y];
        spin_t S_new = S_old + local_alphas_d[x + L*y] * (devRand[idx + offset*N / 2 + N*SWEEPS_EMPTY] - 0.5f);
        //spin_t S_new = S_old + (energy_t) (2.0f * M_PI) * (devRand[idx + offset*N / 2 + N*SWEEPS_EMPTY] - 0.5f);
        S_new = (S_new < 0.0f) ? 0.0f : S_new;
        S_new = (S_new > 2.0f * M_PI) ? 2.0f * M_PI : S_new;

        energy_t E1 = 0.0, E2 = 0.0;

        // NOTE: open boundary conditions -> energy contribution on boundary always results in -cos(S(x,y) - S(x,y)) = -1 
#ifdef DOUBLE_PRECISION
        E1 -= (x == 0) ? 1 : cos(Qfactor * (S_old - s[x - 1 + L*y]));		// from s(x-1,y)
        E2 -= (x == 0) ? 1 : cos(Qfactor * (S_new - s[x - 1 + L*y]));
        E1 -= (x == L - 1) ? 1 : cos(Qfactor * (S_old - s[x + 1 + L*y]));	// from s(x+1,y)
        E2 -= (x == L - 1) ? 1 : cos(Qfactor * (S_new - s[x + 1 + L*y]));
        E1 -= (y == 0) ? 1 : cos(Qfactor * (S_old - s[x + L*(y - 1)]));		// from s(x,y-1)
        E2 -= (y == 0) ? 1 : cos(Qfactor * (S_new - s[x + L*(y - 1)]));
        E1 -= (y == L - 1) ? 1 : cos(Qfactor * (S_old - s[x + L*(y + 1)]));	// from s(x,y+1)
        E2 -= (y == L - 1) ? 1 : cos(Qfactor * (S_new - s[x + L*(y + 1)]));

        if (devRand[idx + offset*N / 2] < exp(-BETA * (E2 - E1)))
        {
            s[x + L*y] = S_new;
            ++Acc;
        }
#else
#ifdef INTRINSIC_FLOAT
        E1 -= (x == 0) ? 1 : __cosf(Qfactor * (S_old - s[x - 1 + L*y]));		// from s(x-1,y)
        E2 -= (x == 0) ? 1 : __cosf(Qfactor * (S_new - s[x - 1 + L*y]));
        E1 -= (x == L - 1) ? 1 : __cosf(Qfactor * (S_old - s[x + 1 + L*y]));	// from s(x+1,y)
        E2 -= (x == L - 1) ? 1 : __cosf(Qfactor * (S_new - s[x + 1 + L*y]));
        E1 -= (y == 0) ? 1 : __cosf(Qfactor * (S_old - s[x + L*(y - 1)]));		// from s(x,y-1)
        E2 -= (y == 0) ? 1 : __cosf(Qfactor * (S_new - s[x + L*(y - 1)]));
        E1 -= (y == L - 1) ? 1 : __cosf(Qfactor * (S_old - s[x + L*(y + 1)]));	// from s(x,y+1)
        E2 -= (y == L - 1) ? 1 : __cosf(Qfactor * (S_new - s[x + L*(y + 1)]));

        if (devRand[idx + offset*N / 2] < __expf(-BETA * (E2 - E1)))
        {
            s[x + L*y] = S_new;
            AccD[x + L*y] = AccD[x + L*y] + 1;
            //if (x + L*y == 35)
                //printf("thread %d local alpha = % 1.5f,  acc_local = %d\n", x + L*y, local_alphas_d[x + L*y], AccD[x + L*y]);
        }
#else
        E1 -= (x == 0) ? 1 : cosf(Qfactor * (S_old - s[x - 1 + L*y]));			// from s(x-1,y)
        E2 -= (x == 0) ? 1 : cosf(Qfactor * (S_new - s[x - 1 + L*y]));
        E1 -= (x == L - 1) ? 1 : cosf(Qfactor * (S_old - s[x + 1 + L*y]));		// from s(x+1,y)
        E2 -= (x == L - 1) ? 1 : cosf(Qfactor * (S_new - s[x + 1 + L*y]));
        E1 -= (y == 0) ? 1 : cosf(Qfactor * (S_old - s[x + L*(y - 1)]));		// from s(x,y-1)
        E2 -= (y == 0) ? 1 : cosf(Qfactor * (S_new - s[x + L*(y - 1)]));
        E1 -= (y == L - 1) ? 1 : cosf(Qfactor * (S_old - s[x + L*(y + 1)]));	// from s(x,y+1)
        E2 -= (y == L - 1) ? 1 : cosf(Qfactor * (S_new - s[x + L*(y + 1)]));

        if (devRand[idx + offset*N / 2] < expf(-BETA * (E2 - E1)))
        {
            s[x + L*y] = S_new;
            ++Acc;
        }
#endif
#endif

    }    
}

__global__ void spin_mult(spin_t *s, spin_t mult_factor)
{
    unsigned int t = threadIdx.x;
    unsigned int b = blockIdx.x;
    unsigned int idx = t + blockDim.x * b;

    s[idx] = s[idx] * mult_factor;
}

__global__ void over_relaxation_k(spin_t *s, spin_t *dilution_mask_d, int offset)
{
    // int t = threadIdx.x + BLOCKL*threadIdx.y;
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = 2 * threadIdx.y + (threadIdx.x + offset) % 2 + BLOCKL*blockIdx.y;

    energy_t sumSin = 0.0, sumCos = 0.0;
    spin_t s_new;

    // checkerboard update
    // not updating spins on the edge of the system
    if (isnan(dilution_mask_d[x + L*y]) && (x > 0) && (x < L - 1) && (y > 0) && (y < L - 1))
    {
        //summation of sin and cos from neighbouring spins
#ifdef DOUBLE_PRECISION
        sumSin += sin(s[x - 1 + L*y]);
        sumCos += cos(s[x - 1 + L*y]);
        sumSin += sin(s[x + 1 + L*y]);
        sumCos += cos(s[x + 1 + L*y]);
        sumSin += sin(s[x + L*(y - 1)]);
        sumCos += cos(s[x + L*(y - 1)]);
        sumSin += sin(s[x + L*(y + 1)]);
        sumCos += cos(s[x + L*(y + 1)]);
#else
#ifdef INTRINSIC_FLOAT
        sumSin += __sinf(s[x - 1 + L*y]);
        sumCos += __cosf(s[x - 1 + L*y]);
        sumSin += __sinf(s[x + 1 + L*y]);
        sumCos += __cosf(s[x + 1 + L*y]);
        sumSin += __sinf(s[x + L*(y - 1)]);
        sumCos += __cosf(s[x + L*(y - 1)]);
        sumSin += __sinf(s[x + L*(y + 1)]);
        sumCos += __cosf(s[x + L*(y + 1)]);
#else
        sumSin += sinf(s[x - 1 + L*y]);
        sumCos += cosf(s[x - 1 + L*y]);
        sumSin += sinf(s[x + 1 + L*y]);
        sumCos += cosf(s[x + 1 + L*y]);
        sumSin += sinf(s[x + L*(y - 1)]);
        sumCos += cosf(s[x + L*(y - 1)]);
        sumSin += sinf(s[x + L*(y + 1)]);
        sumCos += cosf(s[x + L*(y + 1)]);
#endif
#endif
        s_new = (spin_t)(fmod(2.0 * atan2(sumSin, sumCos) - s[x + L*y], 2.0 * M_PI));
        if ((s_new >= 0.0) && (s_new <= Qfactor * 2 * M_PI))
            s[x + L*y] = s_new;
    }
}

__global__ void energyCalc_k(spin_t *s, energy_t *Ed) {

    unsigned int t = threadIdx.x + BLOCKL*threadIdx.y;
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = threadIdx.y + BLOCKL*blockIdx.y;

    energy_t partE = 0;

    // (x,y < L - 1) conditions prevent from accounting bonds outside system boundaries 
#ifdef DOUBLE_PRECISION
    // if (x < L - 1) partE -= cos((energy_t)(Qfactor * (s[x + L*y] - s[x + 1 + L*y])));
    // if (y < L - 1) partE -= cos((energy_t)(Qfactor * (s[x + L*y] - s[x + L*(y + 1)])));
    if (x < L - 1) partE -= cos(Qfactor * (s[x + L*y] - s[x + 1 + L*y]));
    if (y < L - 1) partE -= cos(Qfactor * (s[x + L*y] - s[x + L*(y + 1)]));
#else

#ifdef INTRINSIC_FLOAT
    if (x < L - 1) partE -= __cosf(Qfactor * (s[x + L*y] - s[x + 1 + L*y]));
    if (y < L - 1) partE -= __cosf(Qfactor * (s[x + L*y] - s[x + L*(y + 1)]));
#else
    if (x < L - 1) partE -= cosf(Qfactor * (s[x + L*y] - s[x + 1 + L*y]));
    if (y < L - 1) partE -= cosf(Qfactor * (s[x + L*y] - s[x + L*(y + 1)]));
#endif

#endif

    __shared__ energy_t EnSum[BLOCKL*BLOCKL];
    EnSum[t] = partE;

    for (unsigned int stride = (BLOCKL*BLOCKL) >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (t < stride) EnSum[t] += EnSum[t + stride];
    }

    if (t == 0) Ed[blockIdx.x + gridDim.x*blockIdx.y] = EnSum[0];

}

__global__ void energyCalcDiluted_k(spin_t *s, energy_t *Ed)
{
    unsigned int t = threadIdx.x + BLOCKL*threadIdx.y;
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = threadIdx.y + BLOCKL*blockIdx.y;

    energy_t partE = 0;
    energy_t tryLocalE;


    // (x,y < L - 1) conditions prevent from accounting bonds outside system boundaries 
#ifdef DOUBLE_PRECISION	
    if (x < L - 1)
    {
        tryLocalE = cos(Qfactor * (s[x + L*y] - s[x + 1 + L*y]));
        partE -= isnan(tryLocalE) ? 0 : tryLocalE;
    }
    if (y < L - 1)
    {
        tryLocalE = cos(Qfactor * (s[x + L*y] - s[x + L*(y + 1)]));
        partE -= isnan(tryLocalE) ? 0 : tryLocalE;
    }
#else

#ifdef INTRINSIC_FLOAT
    if (x < L - 1)
    {
        tryLocalE = __cosf(Qfactor * (s[x + L*y] - s[x + 1 + L*y]));
        partE -= isnan(tryLocalE) ? 0 : tryLocalE;
    }
    if (y < L - 1)
    {
        tryLocalE = __cosf(Qfactor * (s[x + L*y] - s[x + L*(y + 1)]));
        partE -= isnan(tryLocalE) ? 0 : tryLocalE;
    }
#else
    if (x < L - 1)
    {
        tryLocalE = cosf(Qfactor * (s[x + L*y] - s[x + 1 + L*y]));
        partE -= isnan(tryLocalE) ? 0 : tryLocalE;
    }
    if (y < L - 1)
    {
        tryLocalE = cosf(Qfactor * (s[x + L*y] - s[x + L*(y + 1)]));
        partE -= isnan(tryLocalE) ? 0 : tryLocalE;
    }
#endif

#endif

    __shared__ energy_t EnSum[BLOCKL*BLOCKL];
    EnSum[t] = partE;

    for (unsigned int stride = (BLOCKL*BLOCKL) >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (t < stride) EnSum[t] += EnSum[t + stride];
    }

    if (t == 0) Ed[blockIdx.x + gridDim.x*blockIdx.y] = EnSum[0];
    //uncomment for verifying the correct simulation temperatures per block
    //verification[x + L*y] = blockIdx.x + gridDim.x*blockIdx.y;
}

__global__ void energyCalcDiluted_per_block(energy_t *Ed, unsigned int *bondCount_d, unsigned int size)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
    {
        Ed[i] = Ed[i] / bondCount_d[i];
        //if(Ed[i] > - 0.4592)
          //  printf("energy in block %d is %1.2f and bond count is %d\n", i, Ed[i], bondCount_d[i]);
    }
}

__global__ void min_max_avg_block(spin_t *d_s, spin_t *d_min, spin_t *d_max, spin_t *d_avg)
{
    unsigned int t = threadIdx.x + BLOCKL*threadIdx.y;
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = threadIdx.y + BLOCKL*blockIdx.y;


    //stores values needed to compute min, max, sum and number of non-NaN values
    __shared__ spin_t min_max_avg_s[4 * BLOCKL*BLOCKL];
    spin_t spin = d_s[x + L*y];
    //if(t == 0)
    //printf("block %d has number = %1.7f\n", blockIdx.x + gridDim.x*blockIdx.y, d_s[x + L*y]);

    min_max_avg_s[t] = spin;
    min_max_avg_s[t + BLOCKL*BLOCKL] = spin;
    min_max_avg_s[t + 2 * BLOCKL*BLOCKL] = isnan(spin) ? 0 : spin;
    min_max_avg_s[t + 3 * BLOCKL*BLOCKL] = isnan(spin) ? 0 : 1;

    for (unsigned int stride = (BLOCKL*BLOCKL) >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (t < stride)
        {
            min_max_avg_s[t] = fmin(min_max_avg_s[t], min_max_avg_s[t + stride]);				// minimum search
            min_max_avg_s[t + BLOCKL*BLOCKL] = fmax(min_max_avg_s[t + BLOCKL*BLOCKL], min_max_avg_s[t + BLOCKL*BLOCKL + stride]);	// maximum search
            min_max_avg_s[t + 2 * BLOCKL*BLOCKL] += min_max_avg_s[t + 2 * BLOCKL*BLOCKL + stride];
            min_max_avg_s[t + 3 * BLOCKL*BLOCKL] += min_max_avg_s[t + 3 * BLOCKL*BLOCKL + stride];
        }
    }

    if (t == 0)
    {
        d_min[blockIdx.x + gridDim.x*blockIdx.y] = min_max_avg_s[0];
        d_max[blockIdx.x + gridDim.x*blockIdx.y] = min_max_avg_s[BLOCKL*BLOCKL];
        d_avg[blockIdx.x + gridDim.x*blockIdx.y] = min_max_avg_s[2 * BLOCKL*BLOCKL] / min_max_avg_s[3 * BLOCKL*BLOCKL];
        //printf("block %d has number = %1.7f\n", blockIdx.x + gridDim.x*blockIdx.y, d_pointers_to_blocks[blockIdx.x + gridDim.x*blockIdx.y][t]);

        //uncomment for verification
        //if(min_max_avg_s[BLOCKL*BLOCKL] > 6.0)
        //printf("block %d has min = %1.7f and max = %1.7f\n", blockIdx.x + gridDim.x*blockIdx.y, min_max_avg_s[0], min_max_avg_s[BLOCKL*BLOCKL]);
        //printf("block %d has avg = %1.7f\n", blockIdx.x + gridDim.x*blockIdx.y, avg[blockIdx.x + gridDim.x*blockIdx.y]);

    }
    //uncomment for verification
    /*
    __syncthreads();
    if (blockIdx.x + gridDim.x*blockIdx.y == 179)
    {
    for (int i = 0; i < BLOCKL*BLOCKL; i++)
    {
    __syncthreads();
    if(t == i)
    printf("thread %d has a low = %1.7f and high = %1.7f\n", t, min_max_avg_s[t], min_max_avg_s[t_off]);
    }

    }
    */
}

__global__ void resetAccD_k(energy_t *AccD) {

    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < BLOCKS) AccD[idx] = 0;

}

__global__ void setInitialLocalAlphas(spin_t *local_alphas_d, spin_t *dilution_mask_d)
{
    unsigned int t = threadIdx.x;
    unsigned int b = blockIdx.x;
    unsigned int idx = t + blockDim.x * b;

    if (isnan(dilution_mask_d[idx]))
        local_alphas_d[idx] = (spin_t) 2*M_PI;
    else
        local_alphas_d[idx] = (spin_t) 0.0;
}

__global__ void setBlockAlphas(energy_t *AccD, spin_t *alphas_per_block_d, int iterations, energy_t acc_rate_min, unsigned int* tryD, spin_t* block_min, spin_t* block_max)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < BLOCKS * 2)
    {
        energy_t acc_rate = AccD[idx] / (energy_t)tryD[idx];

        if (alphas_per_block_d[idx] > 0)
        {
            if (acc_rate < acc_rate_min)
            {
                alphas_per_block_d[idx] = (block_max[idx] - block_min[idx]) / (1 + iterations / (spin_t)SLOPE_RESTR_FACTOR);
            }
        }

    }
}

__global__ void setLocalAlphas(unsigned int *AccD, spin_t *local_alphas_d, int iterations, energy_t acc_rate_min)
{
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = threadIdx.y + BLOCKL*blockIdx.y;
    unsigned int idx = x + L*y;
    
    energy_t acc_rate = AccD[idx] / (energy_t)iterations;

        if (local_alphas_d[idx] > 0)
        {
            if (acc_rate < acc_rate_min)
            {
                local_alphas_d[idx] = 2 * M_PI / (1 + iterations / (spin_t)SLOPE_RESTR_FACTOR);
            }
         }    
}

__global__ void min_max_k(source_t *source_d, source_t *min_d, source_t *max_d, bool isDiluted, spin_t *diluted_mask_d)
{
    unsigned int t = threadIdx.x;
    unsigned int b = blockIdx.x;
    unsigned int idx = t + (blockDim.x * 2) * b;

    unsigned int t_off = t + 256;	// shared memory access with offset - it was calculated too many times

    /* By declaring the shared memory buffer as "volatile", the compiler is forced to enforce
    the shared memory write after each stage of the reduction,
    and the implicit data synchronisation between threads within the warp is restored */
    __shared__ volatile source_t min_max_s[512];
    if (isDiluted)
    {
        min_max_s[t] = source_d[idx] * diluted_mask_d[idx];
        min_max_s[t_off] = source_d[idx + 256] * diluted_mask_d[idx + 256];
    }
    else
    {
        min_max_s[t] = source_d[idx];
        min_max_s[t_off] = source_d[idx + 256];
    }

    __syncthreads();

    // divide min_max_avg_s araray to "min" part (indices 0 ... 255) and "max" part (256 ... 511)
    // macros min(a,b) (and max(a,b)) from math.h are equivalent to conditional ((a < b) ? (a) : (b)) -> will be added in preprocessing
    source_t temp = fmax(min_max_s[t], min_max_s[t_off]);
    min_max_s[t] = fmin(min_max_s[t], min_max_s[t_off]);
    min_max_s[t_off] = temp;

    // unrolling for loop -> to remove instrunction overhead
    __syncthreads();
    if (t < 128)
    {
        min_max_s[t] = fmin(min_max_s[t], min_max_s[t + 128]);				// minimum search
        min_max_s[t_off] = fmax(min_max_s[t_off], min_max_s[t_off + 128]);	// maximum search
    }

    __syncthreads();
    if (t < 64)
    {
        min_max_s[t] = fmin(min_max_s[t], min_max_s[t + 64]);				// minimum search
        min_max_s[t_off] = fmax(min_max_s[t_off], min_max_s[t_off + 64]);	// maximum search
    }

    /* when we have one warp left ->
    no need for "if(t<stride)" and "__syncthreads"
    (no extra work is saved and because instructions are SIMD synchronous within a warp)	*/
    __syncthreads();
    if (t < 32)
    {
        min_max_s[t] = fmin(min_max_s[t], min_max_s[t + 32]);
        min_max_s[t_off] = fmax(min_max_s[t_off], min_max_s[t_off + 32]);

        min_max_s[t] = fmin(min_max_s[t], min_max_s[t + 16]);
        min_max_s[t_off] = fmax(min_max_s[t_off], min_max_s[t_off + 16]);

        min_max_s[t] = fmin(min_max_s[t], min_max_s[t + 8]);
        min_max_s[t_off] = fmax(min_max_s[t_off], min_max_s[t_off + 8]);

        min_max_s[t] = fmin(min_max_s[t], min_max_s[t + 4]);
        min_max_s[t_off] = fmax(min_max_s[t_off], min_max_s[t_off + 4]);

        min_max_s[t] = fmin(min_max_s[t], min_max_s[t + 2]);
        min_max_s[t_off] = fmax(min_max_s[t_off], min_max_s[t_off + 2]);

        min_max_s[t] = fmin(min_max_s[t], min_max_s[t + 1]);
        min_max_s[t_off] = fmax(min_max_s[t_off], min_max_s[t_off + 1]);
    }

    // per block results are stored to global memory
    if (t == 0)
    {
        min_d[b] = min_max_s[0];
        max_d[b] = min_max_s[256];
        //if(min_max_avg_s[BLOCKL*BLOCKL] > 6.0)
        //printf("block %d has min = %1.7f and max = %1.7f\n", b, min_d[b], max_d[b]);
    }
}

__global__ void XY_mapping_k(source_t *source_d, spin_t *XY_mapped_d, source_t minSource, source_t maxSource, bool isDiluted, spin_t *diluted_mask_d)
{
    unsigned int t = threadIdx.x;
    unsigned int b = blockIdx.x;
    unsigned int idx = t + blockDim.x * b;

    XY_mapped_d[idx] = (isDiluted) ? (spin_t)(2 * M_PI * (source_d[idx] * diluted_mask_d[idx] - minSource) / (maxSource - minSource)) :
        (spin_t)(2 * M_PI * (source_d[idx] - minSource) / (maxSource - minSource));
    //if (XY_mapped_d[idx]  > 6.29)
    //printf("block %d has value = %1.7f, maxSource = %1.7f, source_d[%d] = %1.7f\n", b, XY_mapped_d[idx], maxSource, idx, source_d[idx]);

}

__global__ void create_dilution_mask_k(spin_t *dilution_mask_d, float* devRandDil, unsigned int* remSum_d)
{
    unsigned int t = threadIdx.x;
    unsigned int b = blockIdx.x;
    unsigned int idx = t + blockDim.x * b;
    unsigned int rem;
    if (devRandDil[idx] < RemovedDataRatio)
    {
#ifdef DOUBLE_PRECISION
        dilution_mask_d[idx] = nan("");
#else
        dilution_mask_d[idx] = nanf("");
#endif
        rem = 1;
    }
    else
    {
        dilution_mask_d[idx] = 1;
        rem = 0;
    }
    volatile __shared__ unsigned int removed_Sum[256];
    removed_Sum[t] = rem;
    // unrolling for loop -> to remove instrunction overhead
    __syncthreads();
    if (t < 128) removed_Sum[t] += removed_Sum[t + 128];

    __syncthreads();
    if (t < 64) removed_Sum[t] += removed_Sum[t + 64];

    // reduction for last warp
    __syncthreads();
    if (t < 32)
    {
        removed_Sum[t] += removed_Sum[t + 32];
        removed_Sum[t] += removed_Sum[t + 16];
        removed_Sum[t] += removed_Sum[t + 8];
        removed_Sum[t] += removed_Sum[t + 4];
        removed_Sum[t] += removed_Sum[t + 2];
        removed_Sum[t] += removed_Sum[t + 1];
    }

    if (t == 0) remSum_d[b] = removed_Sum[0];
}

__global__ void fill_lattice_nans_averaged(spin_t *XY_mapped_d, spin_t *avg)
{
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = threadIdx.y + BLOCKL*blockIdx.y;
    unsigned int idx = x + L*y;
    if (isnan(XY_mapped_d[idx])) XY_mapped_d[idx] = avg[blockIdx.x + gridDim.x*blockIdx.y];
}

__global__ void fill_lattice_nans_averaged_global(spin_t *XY_mapped_d, spin_t avg)
{
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = threadIdx.y + BLOCKL*blockIdx.y;
    unsigned int idx = x + L*y;
    if (isnan(XY_mapped_d[idx])) XY_mapped_d[idx] = avg;
}

__global__ void fill_lattice_nans_random(spin_t *XY_mapped_d, float*devRand_fill)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (isnan(XY_mapped_d[idx])) XY_mapped_d[idx] = 2 * M_PI * devRand_fill[idx];
}

__global__ void data_reconstruction_k(source_t *reconstructed_d, spin_t *XY_mapped_d, source_t minSource, source_t maxSource, source_t *sum_d, source_t *sumSqr_d)
{
    unsigned int t = threadIdx.x;
    unsigned int b = blockIdx.x;
    unsigned int idx = t + blockDim.x * b;

    reconstructed_d[idx] = ((source_t)XY_mapped_d[idx])*(maxSource - minSource) / (2 * M_PI) + minSource;
    sum_d[idx] += reconstructed_d[idx];
    sumSqr_d[idx] += reconstructed_d[idx] * reconstructed_d[idx];
}

__global__ void bondCount_k(spin_t *mask_d, unsigned int *bondCount_d)
{
    unsigned int t = threadIdx.x + BLOCKL*threadIdx.y;
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = threadIdx.y + BLOCKL*blockIdx.y;

    unsigned int bondCount = 0;
    bool isNotCentralNAN = !isnan(mask_d[x + L*y]);

    if (x < L - 1)
        bondCount += isNotCentralNAN && (!isnan(mask_d[x + 1 + L*y]));
    if (y < L - 1)
        bondCount += isNotCentralNAN && (!isnan(mask_d[x + L*(y + 1)]));

    __shared__ unsigned int bondSum[BLOCKL*BLOCKL];
    bondSum[t] = bondCount;

    for (unsigned int stride = (BLOCKL*BLOCKL) >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (t < stride) bondSum[t] += bondSum[t + stride];
    }

    if (t == 0) bondCount_d[blockIdx.x + gridDim.x*blockIdx.y] = bondSum[0];
    //if (t == 0 && blockIdx.x + gridDim.x*blockIdx.y == 32) bondCount_d[blockIdx.x + gridDim.x*blockIdx.y] = 0;
    //if (t == 0 && bondSum[0] < 20) printf("bond count in block %d is %d\n", blockIdx.x + gridDim.x*blockIdx.y, bondSum[0]);
}

__global__ void mean_stdDev_reconstructed_k(source_t *mean_d, source_t *stdDev_d, unsigned int size)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    mean_d[idx] /= size;
    stdDev_d[idx] = sqrt(stdDev_d[idx] / size + mean_d[idx] * mean_d[idx]);
}

__global__ void sum_prediction_errors_k(source_t *source_d, source_t * mean_d, spin_t *dilution_mask_d,
    source_t *AAE_d, source_t *ARE_d, source_t *AARE_d, source_t *RASE_d, source_t* error_map_d, source_t* error_map_block_d)
{
    unsigned int t = threadIdx.x + BLOCKL*threadIdx.y;
    unsigned int x = threadIdx.x + BLOCKL*blockIdx.x;
    unsigned int y = threadIdx.y + BLOCKL*blockIdx.y;
    unsigned int idx = x + L*y;
    
    source_t source = source_d[idx];
    source_t est_error = (source - mean_d[idx]);
    bool isnan_site = isnan(dilution_mask_d[idx]);

    //get error for each particular spin for error map
    error_map_d[idx] += isnan_site * fabs(est_error);

    volatile __shared__ source_t sum_err[BLOCKL*BLOCKL];
    volatile __shared__ unsigned int validSpins[BLOCKL*BLOCKL];
    // AVERAGE ABSOLUTE ERROR
    sum_err[t] = isnan_site * fabs(est_error);
    validSpins[t] = (unsigned int) isnan_site;
    for (unsigned int stride = (BLOCKL*BLOCKL) >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (t < stride)
        {
            sum_err[t] += sum_err[t + stride];
            validSpins[t] += validSpins[t + stride];
        }
    }

    if (t == 0)
    {
        AAE_d[blockIdx.x + gridDim.x*blockIdx.y] = sum_err[0];
        error_map_block_d[blockIdx.x + gridDim.x*blockIdx.y] += sum_err[0] / (source_t)validSpins[0];
    }
    // AVERAGE RELAITVE ERROR
    __syncthreads();
    sum_err[t] = isnan_site * est_error / source;

    for (unsigned int stride = (BLOCKL*BLOCKL) >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (t < stride) sum_err[t] += sum_err[t + stride];
    }
    if (t == 0) ARE_d[blockIdx.x + gridDim.x*blockIdx.y] = sum_err[0];
    // AVERAGE ABSOLUTE RELATIVE ERROR
    __syncthreads();
    sum_err[t] = isnan_site * fabs(est_error) / source;

    for (unsigned int stride = (BLOCKL*BLOCKL) >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (t < stride) sum_err[t] += sum_err[t + stride];
    }
    if (t == 0) AARE_d[blockIdx.x + gridDim.x*blockIdx.y] = sum_err[0];

    // summation for ROOT AVERAGE SQUARED ROOT
    __syncthreads();
    sum_err[t] = isnan_site * est_error * est_error;

    for (unsigned int stride = (BLOCKL*BLOCKL) >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (t < stride) sum_err[t] += sum_err[t + stride];
    }
    if (t == 0) RASE_d[blockIdx.x + gridDim.x*blockIdx.y] = sum_err[0];
}

__global__ void sum_prediction_errors_k(source_t *source_d, source_t * mean_d, spin_t *dilution_mask_d,
    source_t *AAE_d, source_t *ARE_d, source_t *AARE_d, source_t *RASE_d)
{
    unsigned int t = threadIdx.x;
    unsigned int idx = t + blockDim.x * blockIdx.x;
    source_t source = source_d[idx];
    source_t est_error = (source - mean_d[idx]);
    bool isnan_site = isnan(dilution_mask_d[idx]);

    volatile __shared__ source_t sum_err[256];

    // AVERAGE ABSOLUTE ERROR
    sum_err[t] = isnan_site * fabs(est_error);
    // unrolling for loop -> to remove instrunction overhead
    __syncthreads();
    if (t < 128) sum_err[t] += sum_err[t + 128];
    __syncthreads();
    if (t < 64) sum_err[t] += sum_err[t + 64];
    __syncthreads();
    if (t < 32)
    {
        sum_err[t] += sum_err[t + 32];
        sum_err[t] += sum_err[t + 16];
        sum_err[t] += sum_err[t + 8];
        sum_err[t] += sum_err[t + 4];
        sum_err[t] += sum_err[t + 2];
        sum_err[t] += sum_err[t + 1];
    }
    if (t == 0) AAE_d[blockIdx.x] = sum_err[0];

    // AVERAGE RELAITVE ERROR
    __syncthreads();
    sum_err[t] = isnan_site * est_error / source;

    // unrolling for loop -> to remove instrunction overhead
    __syncthreads();
    if (t < 128) sum_err[t] += sum_err[t + 128];
    __syncthreads();
    if (t < 64) sum_err[t] += sum_err[t + 64];
    __syncthreads();
    if (t < 32)
    {
        sum_err[t] += sum_err[t + 32];
        sum_err[t] += sum_err[t + 16];
        sum_err[t] += sum_err[t + 8];
        sum_err[t] += sum_err[t + 4];
        sum_err[t] += sum_err[t + 2];
        sum_err[t] += sum_err[t + 1];
    }
    if (t == 0) ARE_d[blockIdx.x] = sum_err[0];

    // AVERAGE ABSOLUTE RELATIVE ERROR
    __syncthreads();
    sum_err[t] = isnan_site * fabs(est_error) / source;

    // unrolling for loop -> to remove instrunction overhead
    __syncthreads();
    if (t < 128) sum_err[t] += sum_err[t + 128];
    __syncthreads();
    if (t < 64) sum_err[t] += sum_err[t + 64];
    __syncthreads();
    if (t < 32)
    {
        sum_err[t] += sum_err[t + 32];
        sum_err[t] += sum_err[t + 16];
        sum_err[t] += sum_err[t + 8];
        sum_err[t] += sum_err[t + 4];
        sum_err[t] += sum_err[t + 2];
        sum_err[t] += sum_err[t + 1];
    }
    if (t == 0) AARE_d[blockIdx.x] = sum_err[0];

    // summation for ROOT AVERAGE SQUARED ROOT
    __syncthreads();
    sum_err[t] = isnan_site * est_error * est_error;

    // unrolling for loop -> to remove instrunction overhead
    __syncthreads();
    if (t < 128) sum_err[t] += sum_err[t + 128];
    __syncthreads();
    if (t < 64) sum_err[t] += sum_err[t + 64];
    __syncthreads();
    if (t < 32)
    {
        sum_err[t] += sum_err[t + 32];
        sum_err[t] += sum_err[t + 16];
        sum_err[t] += sum_err[t + 8];
        sum_err[t] += sum_err[t + 4];
        sum_err[t] += sum_err[t + 2];
        sum_err[t] += sum_err[t + 1];
    }
    if (t == 0) RASE_d[blockIdx.x] = sum_err[0];
}

__global__ void find_temperature_gpu(energy_t* E_source, double* T_ref, double* E_ref, energy_t* E_result, int size_source, int size_refs)
{
    for (int myIdx = blockIdx.x * blockDim.x + threadIdx.x; myIdx < size_source; myIdx += blockDim.x * gridDim.x)
    {
        energy_t myEnergy = E_source[myIdx];
        int it_E = 0;
        int it_T = 0;

        while ((it_E != size_refs - 1) && (myEnergy < E_ref[it_E]))
        {
            ++it_E;
            ++it_T;
        }
        // linear interpolation
        E_result[myIdx] = (it_E == 0) ? (T_ref[it_T]) : ((T_ref[it_T] - T_ref[it_T - 1]) * (myEnergy - E_ref[it_E]) / (E_ref[it_E] - E_ref[it_E - 1]) + T_ref[it_T]);
        //E_result[myIdx] = 0.000001;
        //printf("v blocku %d teplota je %1.4f \t energia je %1.4f\n", myIdx, E_result[myIdx], myEnergy);
    }
}

__global__ void correctTemps(energy_t *T_diluted_per_block_d, energy_t medianValue)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < BLOCKS * 2)
    {
        if (T_diluted_per_block_d[idx] < 0 || T_diluted_per_block_d[idx] > 900.0)
            T_diluted_per_block_d[idx] = medianValue;

    }
}

__global__ void smoothTemps(energy_t *T_diluted_per_block_d, energy_t *spinTemps_raw_d, energy_t *spinTemps_smooth_d, bool first_iteration)
{
    int x = threadIdx.x + BLOCKL*blockIdx.x;
    int y = threadIdx.y + BLOCKL*blockIdx.y;
    int idx = x + L*y;
    if (first_iteration)
    {
        spinTemps_raw_d[idx] = T_diluted_per_block_d[blockIdx.x + gridDim.x*blockIdx.y];
        //spinTemps_smooth_d[idx] = T_diluted_per_block_d[blockIdx.x + gridDim.x*blockIdx.y];
        __syncthreads();
        spinTemps_smooth_d[idx] = spinTemps_raw_d[idx];
        return;
    }
    int max_offset = 8;
    int x_offset = 0;
    int y_offset = 0;
    energy_t newLocalTemp = (energy_t) 0.0;
    int neighbor_count = 0;
    //int debug_index = 515;

    while ((y < L - y_offset) && (y_offset <= max_offset))
    {
        
        while ((x - x_offset) >= 0 && (x_offset <= max_offset))
        {
            //if(idx == debug_index)
                //printf("susedov %d na indexe %d\n", neighbor_count+1, x - x_offset + L*(y + y_offset));
            newLocalTemp += spinTemps_raw_d[x - x_offset + L*(y + y_offset)];
            neighbor_count++;
            x_offset++;
        }
        x_offset = 1;
        
        while (x < L - x_offset && x_offset <= max_offset)
        {
            //if (idx == debug_index)
                //printf("susedov %d na indexe %d\n", neighbor_count+1, x + x_offset + L*(y + y_offset));
            newLocalTemp += spinTemps_raw_d[x + x_offset + L*(y + y_offset)];
            neighbor_count++;
            x_offset++;
        }
        x_offset = 0;
        
        y_offset++;
    }
    y_offset = 1;
    x_offset = 0;
   
    while (y - y_offset >= 0 && y_offset <= max_offset)
    {  
        while (x - x_offset > 0 && x_offset <= max_offset)
        {
            //if (idx == debug_index)
                //printf("susedov %d na indexe %d\n", neighbor_count+1, x - x_offset + L*(y - y_offset));
            newLocalTemp += spinTemps_raw_d[x - x_offset + L*(y - y_offset)];
            neighbor_count++;
            x_offset++;
        }
        x_offset = 1;
        while (x < L - x_offset && x_offset <= max_offset)
        {
            //if (idx == debug_index)
                //printf("susedov %d na indexe %d\n", neighbor_count + 1, x + x_offset + L*(y - y_offset));
            newLocalTemp += spinTemps_raw_d[x + x_offset + L*(y - y_offset)];
            neighbor_count++;
            x_offset++;
        }
        x_offset = 0;
        y_offset++;
    }
    //if(y == 1)
        //printf("thread %d, x = %d a y = %d ma %d susedov\n", idx, x, y, neighbor_count);
    spinTemps_smooth_d[idx] = newLocalTemp / (energy_t)neighbor_count;
}

std::vector<energy_t> smoothTempsCPU(energy_t *spinTemps_d)
{
    std::vector<energy_t> newTemps;
    newTemps.resize(L*L);
    for (unsigned int x = 0; x < L; x++)
    {
        for (unsigned int y = 0; y < L; y++)
        {
            unsigned int idx = x + L*y;
            energy_t newLocalTemp = spinTemps_d[idx];
            unsigned int neighbor_count = 1;
            if (x < L - 1)
            {
                newLocalTemp += spinTemps_d[x + 1 + L*y];
                neighbor_count++;
            }
            if (y < L - 1)
            {
                newLocalTemp += spinTemps_d[x + L*(y + 1)];
                neighbor_count++;
                if (x > 0)
                {
                    newLocalTemp += spinTemps_d[x - 1 + L*(y + 1)];
                    neighbor_count++;
                }
                if (x < L - 1)
                {
                    newLocalTemp += spinTemps_d[x + 1 + L*(y + 1)];
                    neighbor_count++;
                }
            }
            if (x > 0)
            {
                newLocalTemp += spinTemps_d[x - 1 + L*y];
                neighbor_count++;
            }
            if (y > 0)
            {
                newLocalTemp += spinTemps_d[x + L*(y - 1)];
                neighbor_count++;
                if (x > 0)
                {
                    newLocalTemp += spinTemps_d[x - 1 + L*(y - 1)];
                    neighbor_count++;
                }
                if (x < L - 1)
                {
                    newLocalTemp += spinTemps_d[x + 1 + L*(y - 1)];
                    neighbor_count++;
                }
            }
            newTemps[idx] = newLocalTemp / (energy_t)neighbor_count;
        }

    }
    return newTemps;

}

energy_t cpu_energy(spin_t *s)
{
    // double ie = 0;
    energy_t partE = 0;
    for (int x = 0; x < L; ++x) {
        for (int y = 0; y < L; ++y) {
#ifdef DOUBLE_PRECISION
            if (x < L - 1) partE -= cos(Qfactor * (s[x + L*y] - s[x + 1 + L*y]));
            if (y < L - 1) partE -= cos(Qfactor * (s[x + L*y] - s[x + L*(y + 1)]));
#else
            if (x < L - 1) partE -= cosf(Qfactor * (s[x + L*y] - s[x + 1 + L*y]));
            if (y < L - 1) partE -= cosf(Qfactor * (s[x + L*y] - s[x + L*(y + 1)]));
#endif
        }
    }
    return partE / Nbond;
}

double find_temperature(energy_t E_source, std::vector<double> T_ref, std::vector<double> E_ref)
{
    auto it_E = E_ref.begin();
    auto it_T = T_ref.begin();

    while ((it_E != E_ref.end()) && (E_source < *it_E))
    {
        ++it_E;
        ++it_T;
    }
    // linear interpolation
    return (it_E == E_ref.begin()) ? (*it_T) : ((*it_T - *(it_T - 1)) * (E_source - *it_E) / (*it_E - *(it_E - 1)) + *it_T);
}

// templates
template <class T> T sumPartialSums(T *parSums_d, int length)
{
    std::vector<T> parSums(length);
    CUDAErrChk(hipMemcpy(parSums.data(), parSums_d, length * sizeof(T), hipMemcpyDeviceToHost));
    T sum = 0;
    for (auto i : parSums) sum += i;
    return sum;
}

template <class T> std::vector<T> findMinMax(T *min_d, T *max_d, int length)
{
    std::vector<T> min_h(length);
    std::vector<T> max_h(length);
    CUDAErrChk(hipMemcpy(min_h.data(), min_d, length * sizeof(T), hipMemcpyDeviceToHost));
    CUDAErrChk(hipMemcpy(max_h.data(), max_d, length * sizeof(T), hipMemcpyDeviceToHost));
    /*T min_temp = *(std::min_element(min_h.begin(), min_h.end()));
    T max_temp = *(std::max_element(max_h.begin(), max_h.end()));
    std::vector<T> min_max = { min_temp, max_temp };*/
    std::vector<T> min_max = { min_h.at(0), max_h.at(0) };
    for (auto i : min_h) min_max.at(0) = std::fmin(min_max.at(0), i);
    for (auto i : max_h) min_max.at(1) = std::fmax(min_max.at(1), i);

    /* std::cout << "Block Minimum elements: ";
    for (auto i : min_h) std::cout << i << " ";
    std::cout << "\n"; */

    return min_max;
}

template <class T> T find_median(T *data_d, int length)
{
    std::vector<T> data_h(length);
    CUDAErrChk(hipMemcpy(data_h.data(), data_d, length * sizeof(T), hipMemcpyDeviceToHost));
    // First we sort the array 
    std::sort(data_h.begin(), data_h.end());
    // check for even case 
    if (length % 2 != 0)
        return data_h[length / 2];
    return (data_h[(length - 1) / 2] + data_h[length / 2]) / (T)2.0;
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void BlockSortKernel(energy_t *d_in, energy_t *d_out)
{
    // Specialize BlockLoad, BlockStore, and BlockRadixSort collective types
    typedef hipcub::BlockLoad<
        energy_t, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE> BlockLoadT;
    //typedef BlockLoad<Key, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;

    typedef hipcub::BlockStore<
        energy_t, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_TRANSPOSE> BlockStoreT;
    typedef hipcub::BlockRadixSort<
        energy_t, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;
    // Allocate type-safe, repurposable shared memory for collectives
    __shared__ union {
        typename BlockLoadT::TempStorage       load;
        typename BlockStoreT::TempStorage      store;
        typename BlockRadixSortT::TempStorage  sort;
    } temp_storage;
    // Obtain this block's segment of consecutive keys (blocked across threads)
    energy_t thread_keys[ITEMS_PER_THREAD];
    int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);
    BlockLoadT(temp_storage.load).Load(d_in + block_offset, thread_keys);

    __syncthreads();    // Barrier for smem reuse
                        // Collectively sort the keys
    BlockRadixSortT(temp_storage.sort).Sort(thread_keys);
    __syncthreads();    // Barrier for smem reuse
                        // Store the sorted segment 
    BlockStoreT(temp_storage.store).Store(d_out + block_offset, thread_keys);
}


// cuRAND errors
char* curandGetErrorString(hiprandStatus_t rc)
{
    switch (rc) {
    case HIPRAND_STATUS_SUCCESS:                   return (char*)curanderr[0];
    case HIPRAND_STATUS_VERSION_MISMATCH:          return (char*)curanderr[1];
    case HIPRAND_STATUS_NOT_INITIALIZED:           return (char*)curanderr[2];
    case HIPRAND_STATUS_ALLOCATION_FAILED:         return (char*)curanderr[3];
    case HIPRAND_STATUS_TYPE_ERROR:                return (char*)curanderr[4];
    case HIPRAND_STATUS_OUT_OF_RANGE:              return (char*)curanderr[5];
    case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:       return (char*)curanderr[6];
#if CUDART_VERSION >= 4010 
    case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED: return (char*)curanderr[7];
#endif
    case HIPRAND_STATUS_LAUNCH_FAILURE:            return (char*)curanderr[8];
    case HIPRAND_STATUS_PREEXISTING_FAILURE:       return (char*)curanderr[9];
    case HIPRAND_STATUS_INITIALIZATION_FAILED:     return (char*)curanderr[10];
    case HIPRAND_STATUS_ARCH_MISMATCH:             return (char*)curanderr[11];
    case HIPRAND_STATUS_INTERNAL_ERROR:            return (char*)curanderr[12];
    default:                                      return (char*)curanderr[13];
    }
}